// Compile with:
// nvcc -lcusparse -o gpu_solver gpu_solver.cu

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(err) if(err != hipSuccess) { \
  std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n"; std::exit(EXIT_FAILURE); }
#define CHECK_CUSPARSE(err) if(err != HIPSPARSE_STATUS_SUCCESS) { \
  std::cerr << "cuSPARSE error\n"; std::exit(EXIT_FAILURE); }

using namespace std;

void readCSRData(const char* path, const char* matrix_name, const char* transformed, int** rowPtrM, int** colIdxM, double** valsM, int* rows, int* cols, int* nnzs);

int main(int argc, char* argv[]) {
  if(argc != 2 && argc != 4) {
    cout << "Usage: programName matID\n";
    cout << "Usage: programName path_to_bin_files matrix_name true/false (transformed/not)\n";
    return 0;
  }

  cout << "injecting CSR data\n";
  int* rowPtr; int* colIdx; double* vals;
  int rows, cols, nnzs;
  readCSRData(argv[1], argv[2], argv[3], &rowPtr, &colIdx, &vals, &rows, &cols, &nnzs);


  // Lower triangular CSR matrix
/*  int h_csrRowPtr[] = {0, 1, 3, 4, 6, 7};
  int h_csrColInd[] = {0, 0, 1, 2, 0, 3, 4};
  float h_csrVal[]   = {3, 2, -2, 1, 4, 1, 3};
  int rows = 5, nnzs = 7;*/

  //float h_xRef[] = {1, 1, 1, 1, 1};
//  float h_xRef[rows];
  float* h_xRef = (float*)malloc(rows * sizeof(float));
  memset(h_xRef, 1.0, rows * sizeof(float));
  float h_b[rows];

  // Manually compute b = L * xRef
  for(int i = 0; i < rows; ++i) {
    float sum = 0;
    for(int j = rowPtr[i]; j < rowPtr[i+1]; ++j)
      sum += vals[j] * h_xRef[colIdx[j]];
    h_b[i] = sum;
  }

  // Device memory
  int *d_csrRowPtr, *d_csrColInd;
  float *d_csrVal, *d_b, *d_x;
  CHECK_CUDA(hipMalloc((void**)&d_csrRowPtr, (rows+1)*sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrColInd, nnzs*sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrVal,    nnzs*sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&d_b,         rows*sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&d_x,         rows*sizeof(float)));

  CHECK_CUDA(hipMemcpy(d_csrRowPtr, rowPtr, (rows+1)*sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrColInd, colIdx, nnzs*sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrVal,    vals,    nnzs*sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_b,         h_b,         rows*sizeof(float), hipMemcpyHostToDevice));

  // cuSPARSE handle & descriptor
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX, vecB;
  void* dBuffer = nullptr;
  size_t bufferSize = 0;

  CHECK_CUSPARSE(hipsparseCreate(&handle));

  CHECK_CUSPARSE(hipsparseCreateCsr(&matA, rows, rows, nnzs,
    d_csrRowPtr, d_csrColInd, d_csrVal,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, rows, d_x, HIP_R_32F));
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecB, rows, d_b, HIP_R_32F));

  // Create SpSV info
  hipsparseSpSVDescr_t spsvDescr;
  CHECK_CUSPARSE(hipsparseSpSV_createDescr(&spsvDescr));

  float alpha = 1.0f;
  CHECK_CUSPARSE(hipsparseSpSV_bufferSize(
    handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, matA, vecB, vecX, HIP_R_32F,
    HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, &bufferSize));

  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

  CHECK_CUSPARSE(hipsparseSpSV_analysis(
    handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, matA, vecB, vecX, HIP_R_32F,
    HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer));

  CHECK_CUSPARSE(hipsparseSpSV_solve(
    handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, matA, vecB, vecX, HIP_R_32F,
    HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr));

  float h_x[rows];
  CHECK_CUDA(hipMemcpy(h_x, d_x, rows*sizeof(float), hipMemcpyDeviceToHost));

  std::cout << "Computed solution x:\n";
  for(int i = 0; i < 40; ++i)
    std::cout << h_x[i] << " ";
  std::cout << "\nExpected (xRef):\n";
  for(int i = 0; i < 40; ++i)
    std::cout << h_xRef[i] << " ";
  std::cout << "\n";

  hipFree(d_csrRowPtr); hipFree(d_csrColInd); hipFree(d_csrVal);
  hipFree(d_b); hipFree(d_x); hipFree(dBuffer);
  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX); hipsparseDestroyDnVec(vecB);
  hipsparseSpSV_destroyDescr(spsvDescr);
  hipsparseDestroy(handle);
  return 0;
}

